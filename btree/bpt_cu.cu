#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "bpt.h"
#define BLOCKBASIC 64
#if (DEBUG==1)
#define __debug(X) X
#else
#define __debug(X)
#endif

/*texture<int ,1 , hipReadModeElementType> texturememory;
static __global__ void gpuSearchPosTextureMemory(int key,int *gpu_pos,int size){
    int globalTx = blockIdx.x * blockDim.x + threadIdx.x;

    if ( globalTx<size-1 ){
       if (tex1Dfetch(texturememory, globalTx) == key){
          *gpu_pos = globalTx;
       }
       if (tex1Dfetch(texturememory, globalTx) < key && tex1Dfetch(texturememory, globalTx+1) > key){
          *gpu_pos = globalTx+1;
       }
    }
}*/


static __global__ void gpuSearchPosShmem1(int key,int * gpu_key_arr,int *gpu_pos,int size) {
    int globalTx = blockIdx.x*blockDim.x+threadIdx.x;   //globalTx=row
    if(globalTx<size){
	   if( key >= gpu_key_arr[globalTx] && key < gpu_key_arr[globalTx+1]){
             *gpu_pos = globalTx;
	   }
	}
     
	
	
}

/*
static __global__ void gpuSearchPosShmem1(int key,int* devKey,int* devPos,int size) {
   int i;
   for(i=0; i< size; ++i){
	  int thiskey = devKey[i];
	  if(key < thiskey){
		 devPos[0] = i;
		 return;
	  }
   }

}
*/
static __global__ void gpuSearchPosShmem1EQ(int key,int* devKey,int* devPos,int size) {
    int globalTx = blockIdx.x*blockDim.x+threadIdx.x;   //globalTx=row
    if(globalTx<size){
       if (devKey[globalTx] == key){
		  devPos[0] = globalTx;
       }
	}
	
}


static int* deviceKey;
static int* devicePos;
const static int negative1 = -2;
extern double getDoubleTime();
#define MEASURETIME(X) 
static int searchPosCudaVersion(int key,int *key_arr,int size,int isequal){
    __debug(printf("[%s] [%s] [%d] key:[%d] key_arr:[%p] size:[%d]\n",__FILE__,__FUNCTION__,__LINE__,key,key_arr,size));
	/*if(!isequal){
       if (key_arr[0]>=key){
          return 0;
       }
       if (key_arr[size-1]<key){
          return size;
       }
       else if (key_arr[size-1]==key){
          return size-1;
       }
	}*/

    int totalSize = sizeof(int)*size;
    int pos = 0;
	if(!deviceKey){
       hipMalloc((void**)&deviceKey,(BPT_ORDER)*sizeof(int));

	}
	if(!devicePos){
	   hipMalloc((void**)&devicePos,sizeof(int));
	}
	if(!isequal){
	   if(key < key_arr[0]){
		  return -1;
	   }
	   if(key >= key_arr[size-1]){
		  return size-1;
	   }
	}
    if(hipMemcpy(devicePos,&negative1,sizeof(int),hipMemcpyHostToDevice) != hipSuccess){
	   printf("%d hipMemcpy Failed(%s)\n",__LINE__,hipGetErrorString(hipGetLastError()));
	   exit(0); 
	}
    if(hipSuccess != hipMemcpy(deviceKey, key_arr, totalSize, hipMemcpyHostToDevice)){
	   printf("%d hipMemcpy Failed(%s)\n",__LINE__,hipGetErrorString(hipGetLastError()));
	   exit(0);
	}
    dim3 dimBlock(BLOCKBASIC,1,1);
    dim3 dimGrid(size/BLOCKBASIC+((size % BLOCKBASIC)>0),1);
	if(isequal){
       gpuSearchPosShmem1EQ<<<dimGrid,dimBlock>>>(key,deviceKey,devicePos,size);
	}
	else{
	   gpuSearchPosShmem1<<<dimGrid,dimBlock>>>(key,deviceKey,devicePos,size-1);
	   if(hipGetLastError() != hipSuccess){
		  printf("kernel launch failed\n");
	   }
	}
	if(hipSuccess !=   hipMemcpy(&pos,devicePos,sizeof(int),hipMemcpyDeviceToHost) ){
	   printf("%d hipMemcpy Failed(%s)\n",__LINE__,hipGetErrorString(hipGetLastError()));
	   exit(0);
	}
	//hipFree(devicePos);
	hipFree(deviceKey);
	//devicePos = 0;
	deviceKey = 0;
	if(isequal){
       return pos;
	}
	else{
	   return pos;
	}
}
extern "C" int bpt_node_find_pos_cuda(bpt_node* n,void* key){
   return searchPosCudaVersion((int)key,(int*)n->keys,n->nkeys,0);
}
extern "C" int bpt_node_find_pos_cuda_eq(bpt_node* n,void* key){
   return searchPosCudaVersion((int)key,(int*)n->keys,n->nkeys,1);
}

